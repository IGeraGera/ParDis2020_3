#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "inc/gaussian.h"
#include "inc/iofun.h"
#include "inc/imgfun.h"

/* CUDA Kernels */
/* Multiply each row of the unrolled data with the gauss filter using grid-stride*/
__global__
void gaussianPass(int patchSize, int dataSize, float *gaussFilter,float *data){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < dataSize; i += stride){
	     data[i] = gaussFilter[i%(patchSize*patchSize)] * data[i];
	}
}
/* Fill the matrix with the distances */
__global__
void distanceMatCalc(long int totalPixels, int availablePixels,int outPixelOffset, int patchSize, float *distMat, float *data,float filtSig){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	
	for (long int i = index; i < availablePixels*totalPixels; i += stride){
	     	int data_i = i / totalPixels +  outPixelOffset;
		int data_j = i % totalPixels;
		float tmp = 0.0;
		/* if not ont diagonal */
	       	if (data_i != data_j){
			for(int elem = 0 ; elem <patchSize*patchSize ; elem++){
				float diff = (data[data_i*patchSize*patchSize + elem] - data[data_j*patchSize*patchSize + elem]);
				tmp += diff * diff;
			}
			tmp = exp(-tmp/(filtSig));
		//	tmp = exp(-tmp/(filtSig*filtSig));
		}	
		distMat[i]=tmp;
	}
}	
/* Find sum of rows for the distance matrix and divide each row element with it 
 * Put the max element of each row to the diagonal */
__global__
void distanceMatFinal(long int totalPixels, int availablePixels, int outPixelOffset, float *distMat){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (long int i = index; i < availablePixels; i += stride){
		float sum = 0.0;
		float max = 0.0;
		for (long int j = 0; j < totalPixels; j++){
			float element = distMat[i*totalPixels+j];
			/* Check if data is max */
			if(element>max) max = element;
			/* Add to sum to divide with it */
			sum += element;
		}
		sum += max;
		/* Iterate row again, put max to diagonal and divide with sum */
		for (long int j = 0; j < totalPixels; j++){
			if ((i + outPixelOffset) == j ) distMat[i*totalPixels+j] = max/sum;
			else distMat[i*totalPixels+j] /= sum;
		}
	}
}
/* Vector Matrix Multiplication */
__global__
void vectorMatrixMult(long int totalPixels, int availablePixels, int outPixelOffset, float* matrix,float *vector, float *out){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (long int i = index; i < availablePixels; i += stride){
		float sum =0.0;
		for (long int j = 0; j < totalPixels; j++){
			sum += matrix[i*totalPixels+j]*vector[j];
		}
		out[i+outPixelOffset]=sum;
	}
	
}
/* for each distance matrix row find max and divide  */
int
main(int argc, char *argv[]){
	/* Check arguments if correct*/
	checkArgsNum(2,argc);
	/* Start Timing*/
	struct timeval startTime,endTime;
	gettimeofday(&startTime,NULL);
	/* Check Patch Size */	
	int patchSize = atoi(argv[2]);
	checkPatchSize(patchSize);
	/* Denoise variabeles */
	float patchSig = 5.0/3.0;
	float filtSig = 0.02;
	/* Declare Image variables */	
 	int rawImgSize_i,rawImgSize_j,padImgSize_i,padImgSize_j;
	float *rawimage, *paddedImg,*patchImg,*gFilter;
	/* Get image from input argument */
	rawimage = getImg(argv[1],&rawImgSize_i, &rawImgSize_j);
	//normalizeImage(rawimage,rawImgSize_i,rawImgSize_j);
	/* Pad Image */
	paddedImg = padImg(rawimage,rawImgSize_i,rawImgSize_j,patchSize,&padImgSize_i,&padImgSize_j);
	/* Get patches from image to rows */
	patchImg = unrollPatches(paddedImg,rawImgSize_i,rawImgSize_j,patchSize);
	/* Calculate the gaussian Filter */
	gFilter = gaussFilter(patchSize,patchSig);
	gaussNorm(gFilter,patchSize);
	/* Multiply gaussian filter with data */
	/* Memory for Kernel */
	float *kernel_gFilter, *kernel_data;
	int totalData = rawImgSize_i*rawImgSize_j*patchSize*patchSize;
		/*CUDA code */
		hipMalloc(&kernel_gFilter,patchSize*patchSize*sizeof(float));
		hipMalloc(&kernel_data, totalData*sizeof(float));
		hipMemcpy(kernel_gFilter,gFilter,patchSize*patchSize*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(kernel_data,patchImg,totalData*sizeof(float),hipMemcpyHostToDevice);	
		gaussianPass<<<(totalData+255)/256,256>>>(patchSize,totalData,kernel_gFilter,kernel_data);
	/* Find Distances matrix */
	/* Check if Distances matrix exceeds ?GB of memory */	
	# define memLimit 8
	long int totalPixels =  rawImgSize_i*rawImgSize_j;
	int distMatParts =0, availablePixels=totalPixels, outPixelOffset=0;
	while(1){
		distMatParts++;
		if (availablePixels*totalPixels*sizeof(float) < (long int) memLimit * (1<<10) * (1<<10) * (1<<10))
			break;
		availablePixels = totalPixels / (distMatParts + 1 ) ;
		
	}

	/*Allocate memory for filtered output */
	float *kernel_rawimage, *kernel_filteredimage;
	hipMalloc( &kernel_rawimage,totalPixels*sizeof(float));
	hipMalloc( &kernel_filteredimage,totalPixels*sizeof(float));
	hipMemcpy(kernel_rawimage,rawimage,totalPixels*sizeof(float),hipMemcpyHostToDevice);
	
	float *filteredimage = (float *)malloc(totalPixels*sizeof(float));
	float *kernel_distMat;
	/* DistMat partitioning */
	for (int part = 0; part < distMatParts; part++){
		/* Find how many rows will distMat will have AKA how many pixels to calculate */
		availablePixels = totalPixels / distMatParts;
		/* If is the last part add the remainder */
		if (part==distMatParts-1) availablePixels += totalPixels % distMatParts;
		/*CUDA code */
		/* Allocate distance matrix */
		hipMalloc(&kernel_distMat,(long int)availablePixels*totalPixels*sizeof(float));
		distanceMatCalc<<<(long int)(availablePixels*totalPixels+255)/256,256>>>(totalPixels,availablePixels,outPixelOffset,patchSize,kernel_distMat,kernel_data,filtSig);
	/* Find sum of rows for the distance matrix and divide each row element with it 
	 * Put the max element of each row to the diagonal */
		distanceMatFinal<<<(availablePixels+255)/256,256>>>(totalPixels,availablePixels,outPixelOffset,kernel_distMat);
		vectorMatrixMult<<<(availablePixels+255)/256,256>>>(totalPixels,availablePixels,outPixelOffset,kernel_distMat,kernel_rawimage,kernel_filteredimage);
		outPixelOffset+=availablePixels;
		hipFree(kernel_distMat);
	}
		/* Get image */
		hipMemcpy(filteredimage,kernel_filteredimage,totalPixels*sizeof(float),hipMemcpyDeviceToHost);

		/* Cuda Freeing memory */
		hipFree(kernel_gFilter);
		hipFree(kernel_data);
		hipFree(kernel_rawimage);
		hipFree(kernel_filteredimage);
	/* End Timer*/
	gettimeofday(&endTime,NULL);

	/* Write image */
	writeImg(argv[1],filteredimage,rawImgSize_i,rawImgSize_j);

	/* Free */ 
	free(patchImg);
	free(rawimage);
	free(filteredimage);
	free(gFilter);
	free(paddedImg);
	/* Calculate Timer  */
	double timeInterval = (endTime.tv_sec * 1000000 + endTime.tv_usec) -
		(startTime.tv_sec * 1000000 + startTime.tv_usec);
	printf("Execution Time : %.2f us",timeInterval);

}
